#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define CUDA_CHECK(stmt) do {                              \
    hipError_t err = stmt;                                \
    if (err != hipSuccess) {                              \
      std::cerr << "CUDA failure in " << #stmt << std::endl\
                << hipGetErrorString(err) << std::endl;   \
      throw std::runtime_error(#stmt);                     \
    }                                                      \
  } while(0)

__device__ constexpr uint32_t threads_per_block_forward_stage_one_parent_cuda_kernel() { return 256; }
__device__ constexpr uint32_t threads_per_block_backward_F_cuda_kernel() { return 256; }
__device__ constexpr uint32_t threads_per_block_backward_R_parent_cuda_kernel() { return 256; }

template<typename T>
__global__ void forward_stage_one_parent_cuda_kernel(
              T*        const __restrict__ output,
        const T*        const __restrict__ W,
        const T*        const __restrict__ C,
        const T*        const __restrict__ F,
        const T*        const __restrict__ Y,
        const T*        const __restrict__ R,
        const T*        const __restrict__ radii,
        const uint32_t* const __restrict__ L_out_list,
        const uint32_t* const __restrict__ L_in_list,
        const uint32_t* const __restrict__ u_sizes,
        const uint32_t* const __restrict__ v_sizes,
        const uint32_t* const __restrict__ output_base_offsets,
        const uint32_t* const __restrict__ C_offsets,
        const uint32_t* const __restrict__ F_base_offsets,
        const uint32_t* const __restrict__ R_base_offsets,
        const uint32_t* const __restrict__ ab_p_to_b,
        const uint32_t					   ab_p_size,
        const uint32_t 					   l_in_max_net
){
    const uint32_t l_out_id  = blockIdx.x;
	const uint32_t l_in_id 	 = blockIdx.y;
	const uint32_t l_in_size = gridDim.y;

	const uint32_t l_out = L_out_list[l_out_id];
	const uint32_t l_in  = L_in_list[l_in_id];
	const uint32_t i_size = 2*l_out + 1;
	const uint32_t j_size = 2*l_in + 1;

	const uint32_t u_size = u_sizes[l_out_id]; 	// output multiplicity (for particular l_out)
	const uint32_t v_size = v_sizes[l_in_id]; 	// input multiplicity  (for particular l_in)

    /*
	  Expected order of indices:
	 	 output -> [l_out, u, i, a, b_p]
	 	 W 		-> [l_out, l_in, 2]
	 	 C		-> [l_out, l_in, l, i, j, m]
	 	 F 		-> [l_in, v, j, a] 					- here indexing over the last index would be in order defined by ab_p_to_b, but cardinality is a
	 	 Y 		-> [l, m, a, b_p]
	 	 R      -> [l_out, l_in, l, u, v, a, b_p]
	 */
	// add offsets
		  T* const __restrict__ output_lout	    = output + (output_base_offsets[l_out_id] * ab_p_size);             // base offsets are the same as for gradients
	const T* const __restrict__ W_lout_lin		= W + (l_out_id * l_in_size + l_in_id) * 2;
	const T* const __restrict__ C_lout_lin		= C + C_offsets[l_out_id*l_in_max_net + l_in_id];                   // TODO: change l_in_max_net + 1 or change to cardinality on prev wrapper
	const T* const __restrict__ F_lin			= F + (F_base_offsets[l_in_id] * a_size);
	const T* const __restrict__ R_lout_lin      = R + (R_base_offsets[l_out_id*l_in_size + l_in_id] * ab_p_size);


	const T W_lout_lin_r_zero    = W_lout_lin[0];
	const T W_lout_lin_r_nonzero = W_lout_lin[1];

	const uint32_t l_min = abs((int32_t)l_out - (int32_t)l_in);
	const uint32_t l_max = l_out + l_in;

	const uint32_t threads_per_block = threads_per_block_forward_stage_one_parent_cuda_kernel()();
	const uint32_t uiab_p_size = u_size * i_size * ab_p_size;

	dim3 blocks((uiab_p_size + threads_per_block - 1)/threads_per_block, 2*min(l_out, l_in)+1);

    forward_stage_one_child_cuda_kernel<<<blocks, threads_per_block>>>(output_lout, C_lout_lin, F_lin, Y, R_lout_lin, radii, ab_p_to_b,
            W_lout_lin_r_nonzero, W_lout_lin_r_zero, l_min, l_max, u_size, v_size, ab_p_size, i_size, j_size);
}

template<typename T>
__global__ void forward_stage_one_child_cuda_kernel(
              T*        const __restrict__ output_lout,
        const T*        const __restrict__ C_lout_lin,
        const T*        const __restrict__ F_lin,
        const T*        const __restrict__ Y,
        const T*        const __restrict__ R_lout_lin,
        const T*        const __restrict__ radii,
        const uint32_t* const __restrict__ ab_p_to_b,
        const T 					       W_lout_lin_r_nonzero,
		const T 					       W_lout_lin_r_zero,
		const uint32_t					   l_min,
		const uint32_t					   l_max,
		const uint32_t					   u_size,
		const uint32_t					   v_size,
		const uint32_t					   ab_p_size,
		const uint32_t 					   i_size,
		const uint32_t 					   j_size
){
    const uint32_t uiab_p = threadIdx.x + blockIdx.x * blockDim.x;

    // last block can be incompletely filled, because uvab_p_size is not necessary divisible by set number of threads
	if (blockIdx.x == gridDim.x - 1 && uiab_p >= u_size * i_size * ab_p_size) return;

	// deduce individual indices
	const uint32_t u	= uiab_p / (i_size * ab_p_size);
	const uint32_t i 	= (uiab_p - u * i_size * ab_p_size) / ab_p_size;
	const uint32_t ab_p = uiab_p - u * i_size * ab_p_size - i * ab_p_size;
	const uint32_t b 	= ab_p_to_b[ab_p];                                      // Note: b is not b_p, it is an index of the only corresponding non-zero entry

	const T norm = W_lout_lin_r_nonzero + (T) (radii[ab_p] != 0.) * (W_lout_lin_r_zero - W_lout_lin_r_nonzero);

	T output_lout_u_i_ab_p_addendum = 0;

	for(uint32_t v = 0; v < v_size; ++v){
	    for(uint32_t j = 0; j < j_size; ++j){
	        for(uint32_t l_f = l_min, l_id = 0; l_f <= l_max; ++l_f, ++l_id){
	            for(uint32_t m = 0, m_size = 2*l_f + 1; m < m_size; ++m){
	                // TODO: recollect indices and distribute over for loops, maybe change order of loops
	                output_lout_u_i_ab_p_addendum +=
	                    C_lout_lin[(l_f*l_f - l_min*l_min)*i_size*j_size + i*j_size*m_size + j*m_size + m] *
	                    F_lin[v*j_size*a_size + j*a_size + b] *
	                    Y[l_f*l_f*ab_p] *
	                    R_lout_lin[l_id*u_size*v_size*ab_p_size + u*v_size*ab_p_size + v*ab_p_size + ab_p];
	            }
	        }
	    }
	}

	atomicAdd(&output_lout[uiab_p], norm * output_lout_u_i_ab_p_addendum);
}


template<typename T>
__global__ void backward_R_child_cuda_kernel(
			  T* 	    const __restrict__ output_lout_lin,
		const T* 	    const __restrict__ C_lout_lin,
		const T* 	    const __restrict__ G_lout,
		const T* 	    const __restrict__ F_lin,
		const T* 	    const __restrict__ Y,
		const T* 	    const __restrict__ radii,
		const uint32_t* const __restrict__ ab_p_to_a,
		const uint32_t* const __restrict__ ab_p_to_b,
		const T 					       W_lout_lin_r_nonzero,
		const T 					       W_lout_lin_r_zero,
		const uint32_t					   l_offset,
		const uint32_t					   u_size,
		const uint32_t					   v_size,
		const uint32_t					   ab_p_size,
		const uint32_t					   a_size,
		const uint32_t 					   i_size,
		const uint32_t 					   j_size
){
	const uint32_t uvab_p = threadIdx.x + blockIdx.x * blockDim.x;

	// last block can be incompletely filled, because uvab_p_size is not necessary divisible by set number of threads
	if (blockIdx.x == gridDim.x - 1 && uvab_p >= u_size * v_size * ab_p_size) return;

	const uint32_t l_f    = blockIdx.y + l_offset;
	const uint32_t m_size = 2*l_f + 1;

	// deduce individual indices
	const uint32_t u	= uvab_p / (v_size * ab_p_size);
	const uint32_t v 	= (uvab_p - u * v_size * ab_p_size) / ab_p_size;
	const uint32_t ab_p = uvab_p - u * v_size * ab_p_size - v * ab_p_size ;
	const uint32_t a    = ab_p_to_a[ab_p];
	const uint32_t b 	= ab_p_to_b[ab_p];

	const T norm = W_lout_lin_r_nonzero + (T) (radii[ab_p] != 0.) * (W_lout_lin_r_zero - W_lout_lin_r_nonzero);

	// add offsets
	const T* const __restrict__ C_lout_lin_l 	= C_lout_lin 	+ (i_size * j_size * (l_f*l_f - l_offset*l_offset)); 	// only valid L's, thus index is shifted
	const T* const __restrict__ G_lout_u		= G_lout 		+ (u * i_size * a_size);
	const T* const __restrict__ F_lin_v		    = F_lin 		+ (v * j_size * a_size);
	const T* const __restrict__ Y_l 			= Y 			+ (l_f * l_f * ab_p_size);							// contains values without gaps along L

	// make additions (writes) to register
	T output_lout_lin_l_uvab_p = 0;

    size_t ijm = 0;
	for (size_t i = 0; i < i_size; i++){
		for (size_t j = 0; j < j_size; j++){
			for (size_t m = 0; m < m_size; m++, ijm++){
				// TODO: store repeating values on different levels to reduce number of calls to global memory
				output_lout_lin_l_uvab_p += C_lout_lin[ijm] * G_lout_u[i*a_size + a] * F_lin_v[j*a_size + b] * Y_l[m*ab_p_size + ab_p];
			}
		}
	}

	// write normalized result to global memory
	// blockIdx.y instead of l_f is intentional, we need consequent zero-based index of l here (not actual value)
	output_lout_lin[blockIdx.y * u_size * v_size * ab_p_size + uvab_p] = norm * output_lout_lin_l_uvab_p;
}


template<typename T>
__global__ void backward_R_parent_cuda_kernel(
			  T* 	    const __restrict__ output,		        // placeholder to store gradients
		const T* 	    const __restrict__ W,			        // normalization coefficients
		const T* 	    const __restrict__ C,			        // Clebsch-Gordan coefficients
		const T* 	    const __restrict__ G,			        // gradients coming from next layer
		const T* 	    const __restrict__ F,			        // input features
		const T* 	    const __restrict__ Y,			        // spherical harmonics
		const T* 	    const __restrict__ radii,		        // absolute distances between points and their neighbors
		const uint32_t* const __restrict__ L_out_list,			// output rotational orders
		const uint32_t* const __restrict__ L_in_list,			// input rotational orders
		const uint32_t* const __restrict__ u_sizes,				// output multiplicities
		const uint32_t* const __restrict__ v_sizes,				// input multiplicities
		const uint32_t* const __restrict__ output_base_offsets,	// jump points for indexing output over l_out, l_in
		const uint32_t* const __restrict__ G_base_offsets,		// jump points for indexing G over l_out
		const uint32_t* const __restrict__ C_offsets,			// jump points for indexing C over l_out, l_in
		const uint32_t* const __restrict__ F_base_offsets, 		// jump points for indexing F over l_in
		const uint32_t* const __restrict__ ab_p_to_a,			// map from composite index ab_p to a
		const uint32_t* const __restrict__ ab_p_to_b,			// map from composite index ab_p to the only b holding non-zero value (contraction of the sum along b)
		const uint32_t					   ab_p_size,			// total number of pairs point-neighbor
		const uint32_t					   a_size,				// number of points (atoms)
		const uint32_t 					   l_in_max_net			// maximal value of l_in that is present in C (for selecting offset)
) {
	const uint32_t l_out_id  = blockIdx.x;
	const uint32_t l_in_id 	 = blockIdx.y;
	const uint32_t l_in_size = gridDim.y;

	const uint32_t l_out = L_out_list[l_out_id];
	const uint32_t l_in  = L_in_list[l_in_id];
	const uint32_t i_size = 2*l_out + 1;
	const uint32_t j_size = 2*l_in + 1;

	const uint32_t u_size = u_sizes[l_out_id]; 	// output multiplicity (for particular l_out)
	const uint32_t v_size = v_sizes[l_in_id]; 	// input multiplicity  (for particular l_in)

	/*
	  Expected order of indices:
	 	 output -> [l_out, l_in, l, u, v, a, b_p]
	 	 W 		-> [l_out, l_in, 2]
	 	 C		-> [l_out, l_in, l, i, j, m]
	 	 G 		-> [l_out, u, i, a]
	 	 F 		-> [l_in, v, j, a] 					- here indexing over the last index would be in order defined by ab_p_to_b, but cardinality is a
	 	 Y 		-> [l, m, a, b_p]
	 */
	// add offsets
		  T* const __restrict__ output_lout_lin	= output + (output_base_offsets[l_out_id*l_in_size + l_in_id] * ab_p_size); // base offsets are the same as for R
	const T* const __restrict__ W_lout_lin		= W + (l_out_id * l_in_size + l_in_id) * 2;
	const T* const __restrict__ C_lout_lin		= C + C_offsets[l_out_id*l_in_max_net + l_in_id];                           // TODO: change l_in_max_net + 1 or change to cardinality on prev wrapper
	const T* const __restrict__ G_lout			= G + (G_base_offsets[l_out_id] * a_size);
	const T* const __restrict__ F_lin			= F + (F_base_offsets[l_in_id] * a_size);

	const T W_lout_lin_r_zero    = W_lout_lin[0];
	const T W_lout_lin_r_nonzero = W_lout_lin[1];

	const uint32_t l_offset = abs((int32_t)l_out - (int32_t)l_in);

	const uint32_t threads_per_block = threads_per_block_backward_R_parent_cuda_kernel();
	const uint32_t uvab_p_size = u_size * v_size * ab_p_size;

	dim3 blocks((uvab_p_size + threads_per_block - 1)/threads_per_block, 2*min(l_out, l_in)+1);

	// TODO: for parity we will need to pass additional list with l filters, or maybe recreate get_l_filters_with_parity here
	backward_R_child_cuda_kernel<<<blocks, threads_per_block>>>(output_lout_lin, C_lout_lin, G_lout, F_lin, Y, radii, ab_p_to_a, ab_p_to_b,
			W_lout_lin_r_nonzero, W_lout_lin_r_zero, l_offset, u_size, v_size, ab_p_size, a_size, i_size, j_size);

    /*
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
    */
}




void backward_R_cuda(
        torch::Tensor output,		        // allocated in higher level wrapper
		torch::Tensor W,                    // layer specific, stored in Hub
		torch::Tensor C,					// object is network wide (sampling is layer specific), stored in Hub
		torch::Tensor G,					// passed by pipeline backward pipeline
		torch::Tensor F,					// layer specific, stored in buffer for backward pass during forward pass
		torch::Tensor Y,					// object is network wide (sampling is layer specific), stored in Hub
		torch::Tensor radii,				// network wide, stored in Hub
		torch::Tensor L_out_list,			// layer specific, stored in Hub
		torch::Tensor L_in_list,			// layer specific, stored in Hub
		torch::Tensor u_sizes,				// layer specific, stored in Hub
		torch::Tensor v_sizes,				// layer specific, stored in Hub
		torch::Tensor output_base_offsets,	// network wide, stored in Hub
		torch::Tensor G_base_offsets,		// layer specific, stored in Hub
		torch::Tensor C_offsets,			// network wide, stored in Hub
		torch::Tensor F_base_offsets, 		// layer specific, stored in Hub
		torch::Tensor ab_p_to_a,			// network wide, stored in Hub
		torch::Tensor ab_p_to_b,			// network wide, stored in Hub
		const uint32_t l_in_max_net			// network wide, stored in Hub // TODO: check if it can be deduced from one of tensor shapes
        ) {

    // const uint32_t ab_p_size,            // deduce from ab_p_to_a.size
    // const uint32_t a_size,			    // deduce from features.size

    // TODO: write kernel call
    /*
    const size_t filters    = Ys.size(0);
    const size_t batch_size = radii.size(0);

    const size_t threads_per_block = 32;                                                    // warp size in contemporary GPUs is 32 threads, this variable should be a multiple of warp size
    dim3 numBlocks((batch_size + threads_per_block - 1)/threads_per_block, filters, 1);     // batch_size/threads_per_block is fractional in general case - round it up

    if (radii.dtype() == torch::kFloat64) {
        rsh_cuda_kernel<double><<<numBlocks, threads_per_block>>>(
            (const double*) radii.data_ptr(), (double*) Ys.data_ptr(), batch_size
        );
    }
    else {                                                                                  // check in C++ binding guarantee that data type is either double (float64) or float (float32)
        rsh_cuda_kernel<float><<<numBlocks, threads_per_block>>>(
            (const float*) radii.data_ptr(), (float*) Ys.data_ptr(), batch_size
        );
    }
    */
}
